#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <stdio.h>
#include <exception>
#include <cmath>
#include "Matrix.cuh"

bool isCorrectAnswer(Matrix const& m, float val, float eps = 1e-6)
{
    for (size_t i = 0; i < m.hight(); i++) {
        for (size_t j = 0; j < m.width(); j++) {
            if (std::abs(m.at(i, j) - val) > eps)
                return false;
        }
    }
    return true;
}

int main()
{
    try 
    {
        size_t s =  1 << 10;
        Matrix m1 = Matrix::full(1.f, 2*s, s);
        Matrix m2 = Matrix::full(1.f, s, s);
        Matrix m3 = m1.mul(m2, Matrix::MulMode::INTRINSICS);

        if (isCorrectAnswer(m3, (float)s))
            printf("CORRECT");
        else
            printf("WRONG");
        printf(" ANSWER\n");

        // hipDeviceReset must be called before exiting in order for profiling and
        // tracing tools such as Nsight and Visual Profiler to show complete traces.
        hipError_t cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return 1;
        }
    }
    catch (std::exception& e) 
    {
        fprintf(stderr, e.what());
        return -1;
    }

    return 0;
}